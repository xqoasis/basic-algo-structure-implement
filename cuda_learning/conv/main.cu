#include "hip/hip_runtime.h"
#include "stdio.h"

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
                        {
                            if(err != hipSuccess)
                            {
                                printf("%s in %s at line %d\n",
                                hipGetErrorString(err),
                                file, line);
                                exit(EXIT_FAILURE);
                            }
                        }
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

// 查找线程
int getThreadNum()
{
    hipDeviceProp_t prop;
    int count;//计算gpu

    HANDLE_ERROR(hipGetDeviceCount(&count));
    printf("gpu num %d\n", count);
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    //block 由grid划分
    printf("max grid dimensions: %d, %d, %d)\n",
     prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

//设备上的 + global
__global__ void conv(float *img, float *kernel, float *result, 
    int width, int height, int kernelSize)
    {
        // 每一次处理一个点
        int ti = threadIdx.x;
        int bi = blockIdx.x;
        int id = (bi * blockDim.x + ti); //图上的第几个点
        if(id >= width * height)
        {
            return;
        }
        int row = id / width;
        int col = id % width;
        for(int i = 0; i < kernelSize; ++i)
        {
            for(int j = 0; j < kernelSize; ++j)
            {
                float imgValue = 0;
                //中心点为row，col，kernalSize奇数
                int curRow = row - kernelSize / 2 + i;//在图上的坐标是第几位
                int curCol = col - kernelSize / 2 + j;
                if(curRow < 0 || curCol < 0 || curRow >= height || curCol >= width)
                {}
                else
                {
                    imgValue = img[curRow * width + curCol];
                }
                //kernel[i * kernelSize + j] 是kernal对应的点
                result[id] += kernel[i * kernelSize + j] * imgValue;
            }

        }
    }

int main()
{
    //定义图
    int width = 1920;
    int height = 1080;
    float *img = new float[width * height]; //img float指针对象
    //给graph赋值
    for(int row = 0; row < height; ++row)
    {
        for(int col = 0; col < width; ++col)
        {
            img[col + row * width] = (col + row) % 256;
        }
    }

    int kernelSize = 3;
    float *kernel = new float[kernelSize * kernelSize];
    for(int i = 0; i < kernelSize * kernelSize; ++i)
    {
        //检测竖着的纹路
        //++i性能更高，i++有一步复制操作，java自动优化
        // -1, 0, 1, -1, 0, 1
        kernel[i] = i % kernelSize - 1;
    }

    float *imgGpu;
    float *kernelGpu;
    float *resultGpu;

    //二重指针：c的malloc返回的是地址，cuda有可能错误（显卡驱动和cuda版本不对称：不能分配，一个指针返回的是错误信息-》再加一个指针）
    HANDLE_ERROR(hipMalloc((void**)&imgGpu, width * height * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&kernelGpu, kernelSize * kernelSize * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&resultGpu, width * height * sizeof(float)));

    HANDLE_ERROR(hipMemcpy(imgGpu, img,
     width * height * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(kernelGpu, kernel,
     kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice));
    
    int threadNum = getThreadNum();
    int blockNum = (width * height - 0.5) / threadNum + 1; //-0。5 防止整除

    conv<<<blockNum, threadNum>> >
        (imgGpu, kernelGpu, resultGpu, width, height, kernelSize);

    float *result = new float[width * height];
    HANDLE_ERROR(hipMemcpy(result, resultGpu,
     width * height * sizeof(float), hipMemcpyDeviceToHost));

    // visualization
    printf("img\n");
    for(int row = 0; row < 10; ++row)
    {
        for(int col = 0; col < 10; ++col)
        {
            printf("%2.0f ", img[col + row * width]);
        }
        printf("\n");
    }
    printf("kernel\n");
    for(int row = 0; row < kernelSize; ++row)
    {
        for(int col = 0; col < kernelSize; ++col)
        {
            printf("%2.0f ", kernel[col + row * kernelSize]);
        }
        printf("\n");
    }

    printf("result\n");
    for(int row = 0; row < 10; ++row)
    {
        for(int col = 0; col < 10; ++col)
        {
            printf("%2.0f ", result[col + row * width]);
        }
        printf("\n");
    }


    return 0;
}