#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

__global__ void add(int* a, int* b, int* c, int num)
{
	//一个线程干的事情
	//当前线程的id，可以换成block的id
	int i = threadIdx.x;
	if(i < num)//防止溢出
	{
		c[i] = a[i] + b[i];
	}
}

int main(void)
{
	// init data
	int num = 10;
	int a[num], b[num], c[num];
	int *a_gpu, *b_gpu, *c_gpu;

	for(int i = 0; i < num; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}

	hipMalloc((void **)&a_gpu, num * sizeof(int)); //分配gpu的空间，不能直接分配，需要给指针的指针
	hipMalloc((void **)&b_gpu, num * sizeof(int));
	hipMalloc((void **)&c_gpu, num * sizeof(int));

	// copy data
	hipMemcpy(a_gpu, a, num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b, num * sizeof(int), hipMemcpyHostToDevice);//gpu的指针，原始数据，size， 状态

	// do
	//一块，十个线程（小项目）
	add<<1, num>>(a_gpu, b_gpu, c_gpu, num);

	// get data
	hipMemcpy(c, c_gpu, num * sizeof(int), hipMemcpyDeviceToHost);

	// visualization
	for(int i = 0; i < num; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	return 0;
}