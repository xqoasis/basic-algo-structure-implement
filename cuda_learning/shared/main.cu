
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N   10
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
__global__ void add( int *a, int *b, int *c ) {
    int tid = threadIdx.x;    // this thread handles the data at its thread id
    // __shared__ int sh[num] 不行
    // __shared__ int new sh[num] 不行
    extern __shared__ int sh[]; //declare
    //use another to get the address
    int *x = (int *)sh;
    if (tid >= N)
        return;
    x[tid] = a[tid] + b[tid];
    c[tid] = x[tid];
}

int main( void ) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    add<<<1,N,N>>>( dev_a, dev_b, dev_c );//第三个N为共享内存的大小

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // display the results
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }

    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    return 0;
}
