#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void sum(float *a, float *b)
{
	//thread id
	//all元素求值，放到新的共享列表里面
	//每个block有共享内存
	int tid = threadIdx.x;

	//变量写在share里面
	__shared__ float sData[16];
	sData[tid] = a[tid];//一个线程管的一位数字
	__syncthreads();//一个block的线程等待同步
	for(int i = 8; i > 0; i /= 2)
	{
		if(tid < i)
		{
			sData[tid] = sData[tid] + sData[tid + i];
		}
		__syncthreads();
	}
	if(tid == 0)
	{
		b[0] = sData[0];
	}
}

int main()
{
	//数组求和
	float a[16];
	//define value
	for(int i = 0; i < 16; i++)
	{
		a[i] = i*(i+1);
	}
	float *aGpu;
	hipMalloc((void**)&aGpu, 16 * sizeof(float));
	hipMemcpy(aGpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);

	float *bGpu;
	hipMalloc((void**)&bGpu, 1 * sizeof(float));
	//一个block 16 thread
	sum<<<1, 16>> >(aGpu, bGpu);

	float b[1];
	hipMemcpy(b, bGpu, 1 * sizeof(float), hipMemcpyDeviceToHost);
	printf("b: %f\n",b[0]);
	return 0;
}