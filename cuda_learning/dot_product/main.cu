#include "hip/hip_runtime.h"
#include <stdio.h>

#define LENGTH 16
#define BLOCKNUM 2
#define THREADNUM 4
__global__ void dot_product(float *a, float *b, float* r)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int total_thread_num = THREADNUM * BLOCKNUM;

	__shared__ float sData[THREADNUM]; //sData[0] = a[0] * b[0] + a[8] * b[8] ...
	int global_id = tid + bid * THREADNUM; //总的第几位(从block0开始数)
	sData[tid] = 0;//初始值
	while(global_id < LENGTH)
	{
		sData[tid] += a[global_id] * b[global_id];
		global_id += total_thread_num;//0， 8， 。。。
	}
	__syncthreads();
	for(int i = THREADNUM/2; i > 0; i /= 2)
	{
		if(tid < i)
		{
			//内存连续，s[0] = s[0] + s[4] -> s[0] = s[0] + s[2]
			sData[tid] = sData[tid] + sData[tid + i];
		}
		__syncthreads();
	}
	if(tid == 0)
	{
		//多个block
		r[bid] = sData[0];
	}
}

int main()
{
	float a[LENGTH];
	float b[LENGTH];
	for(int i = 0; i < LENGTH; i++)
	{
		a[i] = i*(i+1);
		b[i] = i*(i-2);
	}
	float *aGpu;
	hipMalloc((void**)&aGpu, LENGTH * sizeof(float));
	hipMemcpy(aGpu, a, LENGTH * sizeof(float), hipMemcpyHostToDevice);

	float *bGpu;
	hipMalloc((void**)&bGpu, LENGTH * sizeof(float));
	hipMemcpy(bGpu, b, LENGTH * sizeof(float), hipMemcpyHostToDevice);

	float *rGpu;
	hipMalloc((void**)&rGpu, BLOCKNUM * sizeof(float));
	dot_product<<<BLOCKNUM, LENGTH>> >(aGpu, bGpu, rGpu);

	float r[BLOCKNUM];
	hipMemcpy(r, rGpu, BLOCKNUM * sizeof(float), hipMemcpyDeviceToHost);

	float result = 0;
	for(int i = 0; i < BLOCKNUM; i++)
	{
		result += r[i];
	}
	printf("result: %f\n",result);
	return 0;
}