#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <vector>
struct saxpy_functor
{
  const float a;

  saxpy_functor(float _a) : a(_a) {}

  __host__ __device__ //开始计算
  float operator()(const float& x, const float& y) const
  { 
	//自定义
    return y * x + a;
  }
};
//两次读一次写，自定义
void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
  // Y <- A * X + Y
  thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
}

//slow: 4n reads 3n write
void saxpy_slow(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
  thrust::device_vector<float> temp(X.size());
   
  // temp <- A
  thrust::fill(temp.begin(), temp.end(), A);
    
  // temp <- A * X
  // temp和x长度一样，多线程，每个线程xi * tempi
  thrust::transform(X.begin(), X.end(), temp.begin(), temp.begin(), thrust::multiplies<float>());

  // Y <- A * X + Y
  thrust::transform(temp.begin(), temp.end(), Y.begin(), Y.begin(), thrust::plus<float>());
}

int main(void)
{
	thrust::host_vector<float> H(4);
	std::vector<float> I(4);

	// initialize individual elements
	H[0] = 14;
	H[1] = 20;
	H[2] = 38;
	H[3] = 46;
	  // H.size() returns the size of vector H
	std::cout << "H has size " << H.size() << std::endl;

	// print contents of H
	for(int i = 0; i < H.size(); i++)
	{
		std::cout << "H[" << i << "] = " << H[i] << std::endl;
	}
	// Copy host_vector H to device_vector D
	thrust::device_vector<float> D = H; //这里不是共享（内存映射）
	// print contents of D
	for(int i = 0; i < D.size(); i++)
	{
		std::cout << "D[" << i << "] = " << D[i] << std::endl;
	}
	// elements of D can be modified
	D[0] = 99;
	D[1] = 88;
	std::cout<< "D values are changed!" << std::endl;
	for(int i = 0; i < D.size(); i++)
	{
		std::cout << "D[" << i << "] = " << D[i] << std::endl;
	}
	// print contents of H
	for(int i = 0; i < H.size(); i++)
	{
		std::cout << "H[" << i << "] = " << H[i] << std::endl;
	}
	// resize H
  	H.resize(5);
	std::cout << "H now has size " << H.size() << std::endl;
	// set the elements of H to 0, 1, 2, 3, ...
	thrust::sequence(H.begin(), H.end());
		// print contents of H
	for(int i = 0; i < H.size(); i++)
	{
		std::cout << "H[" << i << "] = " << H[i] << std::endl;
	}
	thrust::fill(D.begin(), D.begin() + 3, 9);//还有多种赋值方法
	for(int i = 0; i < D.size(); i++)
	{
		std::cout << "D[" << i << "] = " << D[i] << std::endl;
	}
	thrust::device_vector<float> Y(4);
	// compute Y = -X
  	thrust::transform(D.begin(), D.end(), Y.begin(), thrust::negate<int>());
	for(int i = 0; i < Y.size(); i++)
	{
		std::cout << "Y[" << i << "] = " << Y[i] << std::endl;
	}

	saxpy_fast(2.5, D, Y);
	for(int i = 0; i < Y.size(); i++)
	{
		std::cout << "Y[" << i << "] = " << Y[i] << std::endl;
	}
	return 0;

}